#include "hip/hip_runtime.h"
/* Copyright 2018 Stanford University
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "legion.h"

#include "native_kernels.h"

#include <stdint.h>
#include <inttypes.h>

using namespace Legion;


__global__
void gpu_sum_kernel(Rect<1> rect,
                    const FieldAccessor<READ_ONLY, int8_t, 1, coord_t, Realm::AffineAccessor<int8_t, 1, coord_t> > x,
                    unsigned long long *result)
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const Point<1> p(rect.lo.x + idx);

  // WARNING: This kernel is really, really inefficient. Please don't
  // use this in any context where performance is important!!!

  // FIXME: CUDA only supports atomicAdd on unsigned. Hopefully this
  // cast does sign extension???
  unsigned long long value = x[p];
  atomicAdd(result, value);
}

__host__
int64_t gpu_sum_task(const Task *task,
                     const std::vector<PhysicalRegion> &regions,
                     Context ctx, Runtime *runtime)
{
  assert(regions.size() == 1);

  const FieldAccessor<READ_ONLY, int8_t, 1, coord_t, Realm::AffineAccessor<int8_t, 1, coord_t> > x(regions[0], X_FIELD_ID);

  Rect<1> rect = runtime->get_index_space_domain(ctx,
                  regions[0].get_logical_region().get_index_space());

  const dim3 block(256, 1, 1);
  const dim3 grid(((rect.hi.x - rect.lo.x + 1) + (block.x-1)) / block.x, 1, 1);

  unsigned long long result = 0;

  unsigned long long *gpu_result;
  if (hipMalloc(&gpu_result, sizeof(unsigned long long)) != hipSuccess) {
    abort();
  }

  if (hipMemcpy(gpu_result, &result, sizeof(unsigned long long), hipMemcpyHostToDevice) != hipSuccess) {
    abort();
  }

  gpu_sum_kernel<<<grid, block>>>(rect, x, gpu_result);

  if (hipMemcpy(&result, gpu_result, sizeof(unsigned long long), hipMemcpyDeviceToHost) != hipSuccess) {
    abort();
  }

  int64_t sum = result;
  printf("gpu sum is %" PRId64 "\n", sum);
  return sum;
}
